#include "hip/hip_runtime.h"
#include <stdio.h>
#include "warp/warp/native/builtin.h"
#include "wp_float_arrays_add.cu"
// extern "C" __global__ void add_float_arrays_cuda_kernel_forward(
//     wp::launch_bounds_t dim,
//     wp::array_t<wp::float32> var_dest,
//     wp::array_t<wp::float32> var_a,
//     wp::array_t<wp::float32> var_b);

__global__
void my_kernel(float* dest, float* a, float* b)
{
    int num_items = 8;
    wp::launch_bounds_t bounds_cuda;
    bounds_cuda.ndim = 1;
    bounds_cuda.shape[0] = num_items;
    bounds_cuda.size = num_items;

    wp::array_t<float> var_dest, var_a, var_b;

    var_dest.ndim = 1;
    var_dest.shape[0] = num_items;
    var_dest.strides[0] = sizeof(float);
    var_dest.data = dest;

    var_a.ndim = 1;
    var_a.shape[0] = num_items;
    var_a.strides[0] = sizeof(float);
    var_a.data = a;

    var_b.ndim = 1;
    var_b.shape[0] = num_items;
    var_b.strides[0] = sizeof(float);
    var_b.data = b;

    add_float_arrays_cuda_kernel_forward<<<1,1>>>(bounds_cuda, var_dest, var_a, var_b);
}

int main(void)
{
    float a[8] = {1.1,2.2,3.3,4.4,5.5,6.6,7.7,8.8};
    float b[8] = {100.,200.,300.,400.,500.,600.,700.,800.};
    float dest[8] = {-1.,-1.,-1.,-1.,-1.,-1.,-1.,-1.};

    float* cumem_a;
    float* cumem_b;
    float* cumem_dest;
    hipMalloc((void**)&cumem_a, sizeof(float) * 8);
    hipMalloc((void**)&cumem_b, sizeof(float) * 8);
    hipMalloc((void**)&cumem_dest, sizeof(float) * 8);
    hipMemcpy(cumem_a, a, sizeof(float) * 8, hipMemcpyHostToDevice);
    hipMemcpy(cumem_b, b, sizeof(float) * 8, hipMemcpyHostToDevice);
    hipMemcpy(cumem_dest, dest, sizeof(float) * 8, hipMemcpyHostToDevice);

    my_kernel<<<1, 1>>>(cumem_dest, cumem_a, cumem_b);
    hipMemcpy(dest, cumem_dest, sizeof(float) * 8, hipMemcpyDeviceToHost);
    printf("%f %f %f %f %f %f %f %f\n", dest[0], dest[1], dest[2], dest[3], dest[4], dest[5], dest[6], dest[7]);

    return 0;
}
// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o memset.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -o call.out --device-link wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu --verbose

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -o call_ptx_from_device.o -c /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -Xptxas="-v" -dlink /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu


// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin call_ptx_from_device.o

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// You should be able to run ptx code directly from the cuda driver api with hipModuleLoadDataEx. There is an example here at page 5

// fatbinary --create="/rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin" --image3=kind=ptx,file=wp___main__.sm70.ptx,sm=70


// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu
// ptxas --gpu-name sm_70 --verbose --output-file wp_ptxas_compile.ptx /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.sm70.ptx

// gcc "call_ptx_from_device.cu" -o "call_ptx_from_device.cpp1.ii" 

// nvcc -dryrun call_ptx_from_device.cu  -rdc=true -lcudadevrt
// nvcc call_ptx_from_device.cu  -rdc=true -lcudadevrt