
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" __device__ void add_float_arrays_cuda_kernel_forward(float* a, float* b, float* c);


__global__
void my_kernel(float* a, float* b, float* c)
{
    add_float_arrays_cuda_kernel_forward(a, b, c);
}


int main(void)
{
    float a[4] = {1, 2, 3, 4};
    float b[4] = {1, 2, 3, 4};
    float c[4] = {0, 0, 0, 0};
    // my_kernel<<<1, 1>>>(a, b, c);
    // add_float_arrays_cuda_kernel_forward(c, a, b);
    printf("%f %f %f %f\n", c[0], c[1], c[2], c[3]);

    return 0;
}
// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o memset.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -o call.out --device-link wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu --verbose

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -o call_ptx_from_device.o -c /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -Xptxas="-v" -dlink /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu


// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin call_ptx_from_device.o

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// You should be able to run ptx code directly from the cuda driver api with cuModuleLoadDataEx. There is an example here at page 5