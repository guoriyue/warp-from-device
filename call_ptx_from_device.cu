#include "hip/hip_runtime.h"
#include <stdio.h>
#include "/home/eecs/zhen/.cache/warp/0.11.0/bin/wp___main__.sm70.ptx"

extern "C" __device__ void add_float_arrays_cuda_kernel_forward(float* a, float* b, float* c);


__global__
void my_kernel(float* a, float* b, float* c)
{
    add_float_arrays_cuda_kernel_forward(a, b, c);
}


int main(void)
{
    float a[4] = {1, 2, 3, 4};
    float b[4] = {1, 2, 3, 4};
    float c[4] = {0, 0, 0, 0};
    my_kernel<<<1, 1>>>(a, b, c);

    return 0;
}

// nvcc -o call.out --device-link wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu --verbose