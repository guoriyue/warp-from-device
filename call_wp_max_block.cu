#include "hip/hip_runtime.h"
#include <stdio.h>
#include "warp/warp/native/builtin.h"
// #include "wp_block_max.cu"

extern "C" __global__ void block_max_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float32> var_result,
    wp::array_t<wp::float32> var_a);

__global__
void my_kernel(float* dest, float* a)
{
    int num_items = 64;
    wp::launch_bounds_t bounds_cuda;
    bounds_cuda.ndim = 1;
    bounds_cuda.shape[0] = num_items;
    bounds_cuda.size = num_items;

    wp::array_t<float> var_dest, var_a;

    var_dest.ndim = 1;
    var_dest.shape[0] = num_items;
    var_dest.strides[0] = sizeof(float);
    var_dest.data = dest;

    var_a.ndim = 1;
    var_a.shape[0] = num_items;
    var_a.strides[0] = sizeof(float);
    var_a.data = a;

    block_max_cuda_kernel_forward<<<1,64>>>(bounds_cuda, var_dest, var_a);
}

int main(void)
{
    int array_size = 64;
    float a[array_size];
    float dest[array_size];

    for (int i = 0; i < array_size; i++)
    {
        a[i] = 1.1 + i;
        dest[i] = -1.;
    }

    float* cumem_a;
    float* cumem_dest;
    hipMalloc((void**)&cumem_a, sizeof(float) * array_size);
    hipMalloc((void**)&cumem_dest, sizeof(float) * array_size);
    hipMemcpy(cumem_a, a, sizeof(float) * array_size, hipMemcpyHostToDevice);
    hipMemcpy(cumem_dest, dest, sizeof(float) * array_size, hipMemcpyHostToDevice);

    my_kernel<<<1, 1>>>(cumem_dest, cumem_a);
    hipMemcpy(dest, cumem_dest, sizeof(float) * 8, hipMemcpyDeviceToHost);
    printf("%f\n", dest[0]);
    return 0;
}
// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o memset.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -o call.out --device-link wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu --verbose

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -o call_ptx_from_device.o -c /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// nvcc -arch=sm_70 -Xptxas="-v" -dlink /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu


// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx.out /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin call_ptx_from_device.o

// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu

// You should be able to run ptx code directly from the cuda driver api with hipModuleLoadDataEx. There is an example here at page 5

// fatbinary --create="/rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.fatbin" --image3=kind=ptx,file=wp___main__.sm70.ptx,sm=70


// nvcc -arch=sm_70 -Xptxas="-v" -dlink -o cptx wp___main__.fatbin /rscratch/zhendong/mfguo/warp/warp_cpp/call_ptx_from_device.cu
// ptxas --gpu-name sm_70 --verbose --output-file wp_ptxas_compile.ptx /rscratch/zhendong/mfguo/warp/warp_cpp/wp___main__.sm70.ptx

// gcc "call_ptx_from_device.cu" -o "call_ptx_from_device.cpp1.ii" 

// nvcc -dryrun call_ptx_from_device.cu  -rdc=true -lcudadevrt
// nvcc call_ptx_from_device.cu  -rdc=true -lcudadevrt