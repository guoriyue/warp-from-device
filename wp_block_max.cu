#include "hip/hip_runtime.h"

#define WP_NO_CRT
#include "warp/warp/native/builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)


// block_max.py:26
static __device__ void cuda_block_max(
    wp::array_t<wp::float32> result,
    wp::array_t<wp::float32> a,
    wp::int32 tid)
{

    __shared__ int sdata[64];

    sdata[tid] = a[tid];
    __syncthreads();
    for (int s = 64 / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (sdata[tid] < sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
            }
        }
        __syncthreads();
    }
    if(tid == 0) 
    {
        result[0] = sdata[0];
    }
    }


// block_max.py:26
static __device__ void adj_cuda_block_max(
    wp::array_t<wp::float32> result,
    wp::array_t<wp::float32> a,
    wp::int32 tid,
    wp::array_t<wp::float32> & adj_result,
    wp::array_t<wp::float32> & adj_a,
    wp::int32 & adj_tid)
{
}



extern "C" __global__ void block_max_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float32> var_result,
    wp::array_t<wp::float32> var_a)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        //---------
        // primal vars
        wp::int32 var_0;
        //---------
        // forward
        // def block_max(result: wp.array(dtype=wp.float32),                                      <L 31>
        // tid = wp.tid()                                                                         <L 33>
        var_0 = builtin_tid1d();
        // cuda_block_max(result, a, tid)                                                         <L 34>
        cuda_block_max(var_result, var_a, var_0);
    }
}

extern "C" __global__ void block_max_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float32> var_result,
    wp::array_t<wp::float32> var_a,
    wp::array_t<wp::float32> adj_result,
    wp::array_t<wp::float32> adj_a)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        //---------
        // primal vars
        wp::int32 var_0;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        //---------
        // forward
        // def block_max(result: wp.array(dtype=wp.float32),                                      <L 31>
        // tid = wp.tid()                                                                         <L 33>
        var_0 = builtin_tid1d();
        // cuda_block_max(result, a, tid)                                                         <L 34>
        cuda_block_max(var_result, var_a, var_0);
        //---------
        // reverse
        adj_cuda_block_max(var_result, var_a, var_0, adj_result, adj_a, adj_0);
        // adj: cuda_block_max(result, a, tid)                                                    <L 34>
        // adj: tid = wp.tid()                                                                    <L 33>
        // adj: def block_max(result: wp.array(dtype=wp.float32),                                 <L 31>
        continue;
    }
}

